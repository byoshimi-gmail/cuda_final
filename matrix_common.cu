#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <cassert>            // assert
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "common/common.h"

// Common cuda functions wrapping matrix operations.

// Create matrix populated with random values in host memory.
// Column major order since this is what cusparse library expects.
void generate_dense_float_matrix(int M, int N, float min_val, float max_val, float **outA)
{
    float* out=(float*)malloc(M*N*sizeof(float));

    assert(max_val > min_val);

    for (int col=0; col<N; col++) {
        for (int row=0; row<M; row++) {
            *(out+row*N+col) = (max_val-min_val)*rand() / 2147483647 + min_val;
        }
    }
    *outA = out;
}

// 2-4 (two out of four elements are non-zero, rowwise)
void generate_2_4_sparse_float_matrix_rowwise(int M, int N, float min_val, float max_val, float **outA)
{
    float* out=(float*)malloc(M*N*sizeof(float));

    int masks[6] = {/*0011*/3, /*0101*/5, /*1001*/9, /*0110*/6, /*1010*/ 10, /*1100*/12}; /* 6 variants*/

    assert(max_val > min_val);

    for (int col=0; col<N; col++) {
        for (int row=0; row<M; row+=4) {
            // Min of at least 2 out of every 4 non-zero

            float rnd = ((float)rand() / 2147483647);
            int pick6 = rnd * 6.0;
            int mask=masks[pick6];

            *(out+row*N+col) = mask&0x01?(max_val-min_val)*rand() / 2147483647 + min_val : 0;
            *(out+(row+1)*N+col) = mask&0x02?(max_val-min_val)*rand() / 2147483647 + min_val : 0;
            *(out+(row+2)*N+col) = mask&0x04?(max_val-min_val)*rand() / 2147483647 + min_val : 0;
            *(out+(row+3)*N+col) = mask&0x08?(max_val-min_val)*rand() / 2147483647 + min_val : 0;

        }
    }
    *outA = out;
}


// 2-4 (two out of four elements are non-zero, columnwise
void generate_2_4_sparse_float_matrix_columnwise(int M, int N, float min_val, float max_val, float **outA)
{
    float* out=(float*)malloc(M*N*sizeof(float));

    int masks[6] = {/*0011*/3, /*0101*/5, /*1001*/9, /*0110*/6, /*1010*/ 10, /*1100*/12}; /* 6 variants*/

    assert(max_val > min_val);

    for (int row=0; row<N; row++) {
        for (int col=0; col<M; col+=4) {
            // Min of at least 2 out of every 4 non-zero

            float rnd = ((float)rand() / 2147483647);
            int pick6 = rnd * 6.0;
            int mask=masks[pick6];

            *(out+row*N+col) = mask&0x01?(max_val-min_val)*rand() / 2147483647 + min_val : 0;
            *(out+row*N+col+1) = mask&0x02?(max_val-min_val)*rand() / 2147483647 + min_val : 0;
            *(out+row*N+col+2) = mask&0x04?(max_val-min_val)*rand() / 2147483647 + min_val : 0;
            *(out+row*N+col+3) = mask&0x08?(max_val-min_val)*rand() / 2147483647 + min_val : 0;

        }
    }
    *outA = out;
}


// Print part of a matrix
void print_matrix(char *name, float *M, int nrows, int ncols, int max_row,
        int max_col)
{
    int row, col;

    printf("Dumping matrix %s: (max_rows=%d, max_cols=%d)\n", name, max_row, max_col);

    for (row = 0; row < max_row; row++)
    {
        for (col = 0; col < max_col; col++)
        {
            printf("%2.2f ", M[row * ncols + col]);
        }
        printf("...\n");
    }
    printf("...\n");
}

void sparseTest(int M, int N, int K) {
    float alpha=1;
    float beta=1;

    float *A = (float*)malloc(M*K*sizeof(float));
    float *B = (float*)malloc(K*N*sizeof(float));
    float *C = (float*)malloc(M*N*sizeof(float));

    generate_2_4_sparse_float_matrix_columnwise(M, N, 0.0, 10.0, &A);
    generate_dense_float_matrix(K, N, -1.0, 1.0, &B);
    C = (float *)malloc(sizeof(float) * M * M);

    // Create the cuSPARSE handle
    hipsparseHandle_t handle = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Allocate device memory for vectors and the dense form of the matrix A
    float *dA, *dB, *dC;
    int *dNumZerosPerRowA;
    int totalANnz;


    CHECK(hipMalloc((void **)&dNumZerosPerRowA, sizeof(int) * M));

    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * K));
    CHECK(hipMalloc((void **)&dB, sizeof(float) * K * M));
    CHECK(hipMalloc((void **)&dC, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dNumZerosPerRowA, sizeof(int) * M));

    // Construct a descriptor of the matrix A
    hipsparseMatDescr_t Adescr = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&Adescr));
    CHECK_CUSPARSE(hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO));

    // Transfer the input vectors and dense matrix A to the device
    CHECK(hipMemcpy(dA, A, sizeof(float) * M * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dB, B, sizeof(float) * K * N, hipMemcpyHostToDevice));
    CHECK(hipMemset(dC, 0x00, sizeof(float) * M * N));

    // Compute the number of non-zero elements in A
    CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, K, Adescr,
                                dA, M, dNumZerosPerRowA, &totalANnz));

    /*
    if (totalANnz != trueANnz)
    {
        fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
                "value: expected %d but got %d\n", trueANnz, totalANnz);
        return 1;
    }
    */

    // Allocate device memory to store the sparse CSR representation of A
    float *dCsrValA;
    int *dCsrRowPtrA;
    int *dCsrColIndA;
    CHECK(hipMalloc((void **)&dCsrValA, sizeof(float) * totalANnz));
    CHECK(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalANnz));

    // Convert A from a dense formatting to a CSR formatting, using the GPU
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;



    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, M, K, totalANnz,
        dCsrRowPtrA, dCsrColIndA, dCsrValA,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    // CHECK_CUSPARSE(hipsparseSdense2csr(handle, M, K, Adescr, dA, M, dNumZerosPerRowA,
    //                                  dCsrValA, dCsrRowPtrA, dCsrColIndA,));

    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, K, N, K, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, M, N, M, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    // Copy the result vector back to the host
    CHECK(hipMemcpy(C, dC, sizeof(float) * M * N, hipMemcpyDeviceToHost));

    char a_matrix_desc[]="A:";
    print_matrix(a_matrix_desc, A, M, K, 8, 8);
    char b_matrix_desc[]="b:";
    print_matrix(b_matrix_desc, B, K, N, 8, 8);
    char c_matrix_desc[]="C:";
    print_matrix(c_matrix_desc, C, M, N, 8, 8);

    free(A);
    free(B);
    free(C);

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    CHECK(hipFree(dNumZerosPerRowA));
    CHECK(hipFree(dCsrValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCsrColIndA));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(Adescr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
}