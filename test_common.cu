#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <unistd.h>
#include "matrix_common.h"

void test_generate_dense_float_matrix() {
  int m=20;
  int n=10;
  float minval=-10.0;
  float maxval=100.0;
  float *aMatrix=(float*)malloc(m*n*sizeof(float));

  generate_dense_float_matrix(m, n, minval, maxval, &aMatrix);
  char name[]="test 20x10 dense matrix";
  print_matrix(name, aMatrix, m, n, 10, 10);
}

void test_generate_2_4_sparse_float_matrix() {
  
  int m=8;
  int n=8;
  float minval=-10.0;
  float maxval=100.0;
  float *aMatrix=(float*)malloc(m*n*sizeof(float));

  generate_2_4_sparse_float_matrix_rowwise(m, n, minval, maxval, &aMatrix);
  char name[]="test 20x10 sparse matrix rowwise";
  print_matrix(name, aMatrix, m, n, 8, 8);

  generate_2_4_sparse_float_matrix_columnwise(m, n, minval, maxval, &aMatrix);
  char name2[]="test 20x10 sparse matrix columnwise";
  print_matrix(name2, aMatrix, m, n, 8, 8);
}

int main(void) {
   // test_generate_dense_float_matrix();

   // test_generate_2_4_sparse_float_matrix();
   printf("# algorithm, M, K, N, iterations, total_runtime(ms), average_runtime(ms)\n");
   int sizes[]={8, 16, 64, 256, 512, 1024, 2048, 4096};
   int runs = 8;

   for (int i=0; i<runs; i++) {
    int dim=sizes[i]-1;
    sparseTest(dim, dim, dim, 1000, false);
    cublasTest(dim, dim, dim, 1000, false);
    dim=sizes[i];
    sparseTest(dim, dim, dim, 1000, false);
    cublasTest(dim, dim, dim, 1000, false);
    dim=sizes[i]+2;
    sparseTest(dim, dim, dim, 1000, false);
    cublasTest(dim, dim, dim, 1000, false);
  }
   /*
   for (int i=0; i<runs; i++) {
    //sparseTest(sizes[i]-2, sizes[i]-2, sizes[i]-2, 1000, false);
    //sparseTest(sizes[i]-1, sizes[i]-1, sizes[i]-1, 1000, false);
    int dim=sizes[i]-1;
    sparseTest(dim, dim, dim, 1000, false);
    cublasTest(dim, dim, dim, 1000, false);
  }
   for (int i=0; i<runs; i++) {
     //sparseTest(sizes[i]-2, sizes[i]-2, sizes[i]-2, 1000, false);
     //sparseTest(sizes[i]-1, sizes[i]-1, sizes[i]-1, 1000, false);
     int dim=sizes[i];
     sparseTest(dim, dim, dim, 1000, false);
     cublasTest(dim, dim, dim, 1000, false);
   }
   for (int i=0; i<runs; i++) {
    //sparseTest(sizes[i]-2, sizes[i]-2, sizes[i]-2, 1000, false);
    //sparseTest(sizes[i]-1, sizes[i]-1, sizes[i]-1, 1000, false);
    int dim=sizes[i]+2;
    sparseTest(dim, dim, dim, 1000, false);
    cublasTest(dim, dim, dim, 1000, false);
  }
    */
}